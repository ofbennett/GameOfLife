#include "hip/hip_runtime.h"
#include "World.h"

World::World(int sizex, int sizey):
day(0),
sizex(sizex),
sizey(sizey),
grid(sizex*sizey),
next_grid(sizex*sizey),
d_grid(sizex*sizey),
d_next_grid(sizex*sizey),
index(sizex*sizey),
alive(1),
dead(0)
{
  for (int i=0;i<sizex*sizey;i++) {
    index[i] = i;
  }
}

void World::Populate(int seed){
  srand(seed);
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) {
        this->SetGrid(x,y,rand()%2);
    }
  }
//thrust::copy(d_grid.begin(), d_grid.end(), grid.begin());
}

void World::PopulateFromArray(aliveness data[],int array_length){
  assert(sizex*sizey == array_length);

  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) {
        this->SetGrid(x,y,data[y + (sizey*x)]);
    }
  }
//thrust::copy(d_grid.begin(), d_grid.end(), grid.begin());
}

void World::WriteHeader(ostream &out, int EndOfDays) const{
  out << sizex << " , " << sizey << " , " << EndOfDays << endl;
}

void World::Record(ostream &out) const{
//  thrust::copy(d_grid.begin(), d_grid.end(), grid.begin());
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) {
       out << this->GetGridVal(x,y) << " , ";
    }
    out << endl;
  }
  out << endl;
}

void World::Update(){

  // for (int x=0;x<sizex;x++) {
  //   for (int y=0;y<sizey;y++) {
  //     this->SetNextGrid(x,y,NewState(x,y));
  //   }
  // }
  // grid = next_grid;

  // UpdateKernel<<<1,64>>>(d_grid,index,d_next_grid,sizex,sizey);
  UpdateKernel(d_grid,index,d_next_grid,sizex,sizey);

//  thrust::copy(d_next_grid.begin(), d_next_grid.end(), d_grid.begin());

  day += 1;
}

int World::Day() const{
  return day;
}

int World::Sizex() const{
  return sizex;
}

int World::Sizey() const{
  return sizey;
}

host_grid_type World::Grid() const{
  return grid;
}

int World::Size() const{
  return sizex*sizey;
}

// aliveness World::NewState(int x, int y) const{
//   int newstate;
//   int alive_neighbors = 0;
//   int xn; // Neighbor x index
//   int yn; // Neighbor y index
//
//   // Count alive neighbors
//   for (int i=0;i<3;i++) {
//     for (int j=0;j<3;j++) {
//       if((i==1)&&(j==1)){continue;}
//
//       xn = x-1+i;
//       yn = y-1+j;
//
//       // Wrap index around to other side of grid if at an edge
//       if((x-1+i)==sizex){xn = 0;}
//       if((y-1+j)==sizey){yn = 0;}
//       if((x-1+i)==-1){xn = sizex-1;}
//       if((y-1+j)==-1){yn = sizey-1;}
//
//       if(this->GetGridVal(xn,yn)==alive){alive_neighbors += 1;}
//     }
//   }
//
//   if(this->GetGridVal(x,y)==alive){
//     if((alive_neighbors==2)||(alive_neighbors==3)){newstate = alive;}
//     else{newstate = dead;}
//   }else if(this->GetGridVal(x,y)==dead){
//     if(alive_neighbors==3){newstate = alive;}
//     else{newstate = dead;}
//   }else{
//     throw logic_error("A lifeform on the grid has an aliveness which is neither alive or dead!");
//   }
//   return newstate;
// }

void World::SetGrid(int x, int y, aliveness val){
  grid[y + x*sizey] = val;
}

void World::SetNextGrid(int x, int y, aliveness val){
  next_grid[y + x*sizey] = val;
}

aliveness World::GetGridVal(int x, int y) const{
  aliveness val = grid[y + x*sizey];
  return val;
}
